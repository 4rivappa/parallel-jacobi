#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

int N = 256;
#define BLOCK_SIZE 16

void load_input_data(double *A, double *B, double *x, int n){
    FILE *f;
    f = fopen("input.txt", "r");
    if (f == NULL) {
        printf("Error opening input file.\n");
        exit(1);
    }
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n-1; j++) {
            fscanf(f, "%lf", &A[i*n + j]);
        }
        fscanf(f, "%lf", &B[i]);
        x[i] = 0.0;
    }
    fclose(f);
}

__global__ void jacobi_kernel(double *u, double *f, double *u_new, double h, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < N-1 && j > 0 && j < N-1)
    {
        int index = i*N + j;
        u_new[index] = 0.25 * (u[index-N] + u[index-1] + u[index+1] + u[index+N] - h*h*f[index]);
    }
}

int jacobi_algorithm(int s){
    // size
    N = s;
    // generating matrix
    char command_str[300];
    sprintf(command_str, "python generate_matrix.py %d input.txt", s);
    system(command_str);

    double *u, *f, *u_new;
    double h = 1.0/(N-1);
    double tol = 1e-6;
    int maxiter = 10000;
    int iter = 0;
    size_t size = N*N*sizeof(double);
    hipEvent_t start, stop;
    float elapsedTime;

    hipMallocManaged(&u, size);
    hipMallocManaged(&f, size);
    hipMallocManaged(&u_new, size);

    // Initialize u and f
    load_input_data(f, u, u_new, N);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N+BLOCK_SIZE-1)/BLOCK_SIZE, (N+BLOCK_SIZE-1)/BLOCK_SIZE);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Jacobi iteration
    while (iter < maxiter)
    {
        jacobi_kernel<<<grid, block>>>(u, f, u_new, h, N);

        hipDeviceSynchronize();

        // Check for convergence
        double diff_norm = 0.0;
        for (int i = 1; i < N-1; i++)
        {
            for (int j = 1; j < N-1; j++)
            {
                int index = i*N + j;
                diff_norm += pow(u_new[index] - u[index], 2);
            }
        }
        diff_norm = sqrt(diff_norm)/(N-2)/(N-2);

        if (diff_norm < tol)
        {
            break;
        }

        // Swap u and u_new
        double *temp = u;
        u = u_new;
        u_new = temp;

        iter++;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time: %f ms\n", elapsedTime);

    FILE *fp;
    fp = fopen("cuda_results.txt", "a");
    if (fp == NULL){
        printf("Error in opening file !!\n");
        return -1;
    }
    fprintf(fp, "cuda %d %f\n", s, elapsedTime);
    fclose(fp);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free memory
    hipFree(u);
    hipFree(f);
    hipFree(u_new);

    return 0;
}

int main(){
    for(int s = 100; s < 1050; s += 50){
        jacobi_algorithm(s);
    }
}